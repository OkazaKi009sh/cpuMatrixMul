
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define MODULO 100

unsigned int seed = (unsigned)time(NULL);

void MatrixMultiplication(float* M, float* N, float* P, int Width) {
	for (int i = 0; i < Width; ++i) {
		for(int j = 0; j < Width; ++j) {
			float sum = 0;
			for(int k = 0; k < Width; ++k) {
				float a = M[i * Width + k];
				float b = N[k * Width + j];
				sum += a * b;
			}
		P[i * Width + j] = sum;	
		}
	}
}
void printMatrix(float* T, int Width) {
	for(int i=0; i < Width; ++i) {
		for(int j=0; j < Width; ++j) {
			printf("%f ",T[i * Width + j]);
		}
		printf("\n");
	}
}
void createRandomMatrix(float* T, int Width) {
	srand(seed++);
	float random = 5;
	for(int i=0; i < Width; ++i) {
		for(int j=0; j < Width; ++j) {
			random = rand() % MODULO + 1;
			T[i * Width + j] = random;
		}
	}
}
int main() {
	const int Width = 100;
	float M[Width*Width],N[Width*Width],P[Width*Width];
	unsigned int start = 0;
	unsigned int end = 0;
	
	createRandomMatrix(M,Width);
	printf("M is ...\n");
	//printMatrix(M,Width);
	createRandomMatrix(N,Width);
	printf("N is ...\n");
	//printMatrix(N,Width);
	start = clock();
	MatrixMultiplication(M,N,P,Width);
	end = clock();
	printf("P is ...\n");
	//printMatrix(P,Width);
	printf("time is %u",end - start);
	return 0;
}